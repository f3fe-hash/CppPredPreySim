#include "hip/hip_runtime.h"
#pragma once

#include <cmath>
#include <vector>

#include <hip/hip_runtime.h>

#include "utils/defs.hpp"

// CUDA kernel for mult_add partial sums (dot product + add scalar)
__global__ void mult_add_kernel(const num* a, const num* b, num c, std::size_t n, num* result)
{
    extern __shared__ num sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    num tmp = 0;

    if (i < n)
        tmp = a[i] * b[i];

    sdata[tid] = tmp;
    __syncthreads();

    // Parallel reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
            sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    if (tid == 0)
        atomicAdd(result, sdata[0]);
}

inline num mult_add(const num_arr& a, const num_arr& b, const num c, std::size_t n) noexcept
{
    // Allocate device memory
    num *d_a, *d_b, *d_result;
    hipMalloc(&d_a, n * sizeof(num));
    hipMalloc(&d_b, n * sizeof(num));
    hipMalloc(&d_result, sizeof(num));

    num zero_val = 0;
    hipMemcpy(d_a, a.data(), n * sizeof(num), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), n * sizeof(num), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &zero_val, sizeof(num), hipMemcpyHostToDevice);

    // Launch kernel
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    size_t sharedMemSize = blockSize * sizeof(num);
    mult_add_kernel<<<gridSize, blockSize, sharedMemSize>>>(d_a, d_b, c, n, d_result);

    // Copy result back
    num gpu_result = 0;
    hipMemcpy(&gpu_result, d_result, sizeof(num), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return gpu_result + c;
}

// CUDA kernel for activation (sigmoid)
__global__ void activation_kernel(const num* input, num* output, std::size_t n)
{
    std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        output[i] = 1 / (1 + expf(-input[i]));
}

inline num_arr activation(const num_arr a, std::size_t n) noexcept
{
    num_arr result(n);
    num *d_input, *d_output;

    hipMalloc(&d_input, n * sizeof(num));
    hipMalloc(&d_output, n * sizeof(num));

    hipMemcpy(d_input, a.data(), n * sizeof(num), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    activation_kernel<<<gridSize, blockSize>>>(d_input, d_output, n);

    hipMemcpy(result.data(), d_output, n * sizeof(num), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return result;
}

// CUDA kernel for activation derivative
__global__ void activation_derv_kernel(const num* input, num* output, std::size_t n)
{
    std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        output[i] = input[i] * (1 - input[i]);
}

inline num_arr activation_derv(const num_arr a, std::size_t n) noexcept
{
    num_arr result(n);
    num *d_input, *d_output;

    hipMalloc(&d_input, n * sizeof(num));
    hipMalloc(&d_output, n * sizeof(num));

    hipMemcpy(d_input, a.data(), n * sizeof(num), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    activation_derv_kernel<<<gridSize, blockSize>>>(d_input, d_output, n);

    hipMemcpy(result.data(), d_output, n * sizeof(num), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return result;
}

// CUDA kernel for error calculation
__global__ void error_kernel(const num* x, const num* y, num* output, std::size_t n)
{
    std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        output[i] = x[i] - y[i];
}

inline num_arr error(const num_arr x, const num_arr y, std::size_t n) noexcept
{
    num_arr result(n);
    num *d_x, *d_y, *d_output;

    hipMalloc(&d_x, n * sizeof(num));
    hipMalloc(&d_y, n * sizeof(num));
    hipMalloc(&d_output, n * sizeof(num));

    hipMemcpy(d_x, x.data(), n * sizeof(num), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y.data(), n * sizeof(num), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    error_kernel<<<gridSize, blockSize>>>(d_x, d_y, d_output, n);

    hipMemcpy(result.data(), d_output, n * sizeof(num), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_output);

    return result;
}